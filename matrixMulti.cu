
#include <hip/hip_runtime.h>
#include <stdio.h>
// Matrix dimensions
// CUDA kernel for matrix multiplication
__global__ void matMulKernel(float *A, float *B, float *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float value = 0.0f;
        for (int k = 0; k < N; k++) {
            value += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = value;
    }
}

void matMul(float *A, float *B, float *C, int N) {
    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, N * N * sizeof(float));
    hipMalloc(&d_B, N * N * sizeof(float));
    hipMalloc(&d_C, N * N * sizeof(float));

    // Copy matrices to device memory
    hipMemcpy(d_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(float), hipMemcpyHostToDevice);

    // Define block size and grid size
    dim3 blockDim(16, 16);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y);

    // Launch the kernel
    matMulKernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, N);

    // Copy the result back to host memory
    hipMemcpy(C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    int N = 1024;

    // Allocate host memory
    float *A = (float*)malloc(N * N * sizeof(float));
    float *B = (float*)malloc(N * N * sizeof(float));
    float *C = (float*)malloc(N * N * sizeof(float));

    

    // Initialize matrices A and B
    for (int i = 0; i < N * N; i++) {
        A[i] = rand() % 100;
        B[i] = rand() % 100;
    }

    // Perform matrix multiplication
    matMul(A, B, C, N);

    // Optionally: Print a portion of the result matrix C
    printf("C[0][0] = %f\n", C[0]);

    // Free host memory
    free(A);
    free(B);
    free(C);

    return 0;
}
